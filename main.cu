

#include <hip/hip_runtime.h>
#include <iostream>

// Imported Functions
extern void adjustSaturation(char *image, float saturation);

int main() {
    char* imageName = "./farmImg.webp";
    float saturationFactor = 1.0f; // Example saturation factor

	adjustSaturation(imageName, saturationFactor);
	printf("Adjustment complete.\n");
	return 0;
}
