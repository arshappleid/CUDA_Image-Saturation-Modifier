#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <stdio.h>


__device__ void rgbToHsv(unsigned char r, unsigned char g, unsigned char b, float *h, float *s, float *v);
__device__ void hsvToRgb(float h, float s, float v, unsigned char *r, unsigned char *g, unsigned char *b);
__global__ void adjustSaturationKernel(unsigned char *image, float saturation, int width, int height, int channels);

void adjustSaturation(char *imageName, float saturationFactor){
	// This is the wrapper function to adjustSaturationKernel

	// Read the image using OpenCV
    cv::Mat img = cv::imread(imageName, cv::IMREAD_COLOR);
    if (img.empty()) {
		printf("Error: Image cannot be loaded.");
		return;
	}
	
	// Convert image to a flat array of unsigned chars for CUDA
	cv::Mat imgFlat = img.reshape(1, img.total() * img.channels());
    unsigned char *image;
    
    // Allocate unified memory accessible by both host and device
    size_t imageSize = img.total() * img.channels();
    hipMallocManaged(&image, imageSize);

    // Copy the image data into the managed memory
    memcpy(image, imgFlat.ptr(), imageSize);

    // Define the block and grid sizes
    dim3 blockSize(16, 16); // You can tune these values
    dim3 gridSize((img.cols + blockSize.x - 1) / blockSize.x, (img.rows + blockSize.y - 1) / blockSize.y);


    adjustSaturationKernel<<<gridSize, blockSize>>>(image, saturationFactor, img.cols, img.rows, img.channels());

    hipDeviceSynchronize();

    cv::Mat resultImg = cv::Mat(img.size(), img.type(), image).clone();
    std::string modifiedImageName = "modified_" + std::string(imageName);
    cv::imwrite(modifiedImageName, resultImg);
    hipFree(image);
}

__global__ void adjustSaturationKernel(unsigned char *image, float saturation, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        
        // Read the pixel
        unsigned char r = image[idx];
        unsigned char g = image[idx + 1];
        unsigned char b = image[idx + 2];
        
        // Convert to HSV
        float h, s, v;
        rgbToHsv(r, g, b, &h, &s, &v);
        
        // Adjust the saturation
        s *= saturation;
        
        // Convert back to RGB
        hsvToRgb(h, s, v, &r, &g, &b);
        
        // Write the pixel back
        image[idx] = r;
        image[idx + 1] = g;
        image[idx + 2] = b;
    }
}

__device__ void rgbToHsv(unsigned char r, unsigned char g, unsigned char b, float *h, float *s, float *v) {
    float red = r / 255.0f;
    float green = g / 255.0f;
    float blue = b / 255.0f;

    float cmax = fmaxf(red, fmaxf(green, blue));
    float cmin = fminf(red, fminf(green, blue));
    float delta = cmax - cmin;

    // Hue calculation
    if (delta == 0) {
        *h = 0;
    } else if (cmax == red) {
        *h = 60.0f * fmodf(((green - blue) / delta), 6.0f);
    } else if (cmax == green) {
        *h = 60.0f * (((blue - red) / delta) + 2.0f);
    } else {
        *h = 60.0f * (((red - green) / delta) + 4.0f);
    }

    // Saturation calculation
    *s = (cmax == 0) ? 0 : (delta / cmax);

    // Value calculation
    *v = cmax;
}

__device__ void hsvToRgb(float h, float s, float v, unsigned char *r, unsigned char *g, unsigned char *b) {
    float c = v * s;
    float x = c * (1 - fabsf(fmodf(h / 60.0f, 2) - 1));
    float m = v - c;
    float r_, g_, b_;

    if (h >= 0 && h < 60) {
        r_ = c, g_ = x, b_ = 0;
    } else if (h >= 60 && h < 120) {
        r_ = x, g_ = c, b_ = 0;
    } else if (h >= 120 && h < 180) {
        r_ = 0, g_ = c, b_ = x;
    } else if (h >= 180 && h < 240) {
        r_ = 0, g_ = x, b_ = c;
    } else if (h >= 240 && h < 300) {
        r_ = x, g_ = 0, b_ = c;
    } else {
        r_ = c, g_ = 0, b_ = x;
    }

    *r = (unsigned char)((r_ + m) * 255.0f);
    *g = (unsigned char)((g_ + m) * 255.0f);
    *b = (unsigned char)((b_ + m) * 255.0f);
}


